#include "hip/hip_runtime.h"

#include "allStepsInParallelKernel.cuh"

__global__ void allStepsInParallelBitonicSort(int* deviceElements, int bitonicSequenceSize, int steps){

	//1printf("Kernal Called!!!!");
	/*
	Here we get our first thread var i and j.
	we get j by knowing the size of the subsequence and then halfing it, this gives us the rang that values should be comapired for this step.
	As we go down the steps, we'll be halfing j until step = 1;
	*/

	int firstIndex = threadIdx.x + blockDim.x * blockIdx.x;
	int rangeOfComparison = (bitonicSequenceSize / 2);
	for (int step = steps; step >= 1; step--){

		//This xor op checks that our second value is bigger than our firstIndex value
		if ((firstIndex ^ rangeOfComparison) > firstIndex){

			//assending
			if ((firstIndex / bitonicSequenceSize) % 2 == 0){

				if (deviceElements[firstIndex] > deviceElements[firstIndex ^ rangeOfComparison]) {
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[firstIndex ^ rangeOfComparison];
					deviceElements[firstIndex ^ rangeOfComparison] = temp;
				}

			}
			else{

				if (deviceElements[firstIndex] < deviceElements[firstIndex ^ rangeOfComparison]) {
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[firstIndex ^ rangeOfComparison];
					deviceElements[firstIndex ^ rangeOfComparison] = temp;
				}

			}

		}
		__syncthreads();

		rangeOfComparison = rangeOfComparison / 2;
	}
}
/*
Main function call. Created array and calls stepskernel based of the size of the bitonic sequences and step.
*/
void allStepsInParallelSetup(int* elements, int elementsToSort, int deviceBlocks, int threadsPerBlock){

	int* d_elements;

	//get "phases" so we know how many times we need to send array over to device  
	int phases = int(log2(double(elementsToSort)));

	//General cuda managment here : Allocate on device, array isn't going to change  in size
	hipMalloc(&d_elements, elementsToSort*sizeof(int));
	hipMemcpy(d_elements, elements, elementsToSort*sizeof(int), hipMemcpyHostToDevice);
	dim3 blocks(deviceBlocks, 1);    /* Number of blocks   */
	dim3 threads(threadsPerBlock, 1);  /* Number of threads  */

	for (int currentPhase = 1; currentPhase <= phases; currentPhase++){

		//Get the  size of each sub sequence and the amount of "Steps" in the individual sub sequences 
		int bitonicSequenceSize = int(pow(double(2), double(currentPhase)));

		int steps = int(log2((double)bitonicSequenceSize));

		hipDeviceSynchronize();
		hipDeviceSynchronize();
		allStepsInParallelBitonicSort << <blocks, threads >> >(d_elements, bitonicSequenceSize, steps);
		hipDeviceSynchronize();
	}
	hipMemcpy(elements, d_elements, elementsToSort*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_elements);
}