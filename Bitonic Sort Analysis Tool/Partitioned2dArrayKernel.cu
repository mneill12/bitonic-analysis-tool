#include "hip/hip_runtime.h"
#include "partitioned2dArrayKernel.cuh"

arrays splitArray(int* origionalElements, arrays evenOddArrays, int elementCount){

	int *evenCountPtr = evenOddArrays.evenArray;
	int *oddCountPtr = evenOddArrays.oddArray;
	int count = 0;
	for (int i = 0; i < elementCount; i++){

		if (i % 2 == 0){

			memcpy(evenCountPtr, origionalElements, sizeof(int));

			evenCountPtr++;
		}
		else{
			memcpy(oddCountPtr, origionalElements, sizeof(int));

			oddCountPtr++;
		}
		origionalElements++;
	}

	return evenOddArrays;
}

//Merger even and odd arrays into origional arrays
void mergeArrays(arrays evenOddArrays, int* origionalElemens, int elementCount){

	int *evenCountPtr = evenOddArrays.evenArray;
	int *oddCountPtr = evenOddArrays.oddArray;
	int count = 0;

	for (int i = 0; i < elementCount; i++){

		if (i % 2 == 0){

			//Destination, Source, size
			memcpy(origionalElemens, evenCountPtr, sizeof(int));
			evenCountPtr++;
		}
		else{
			memcpy(origionalElemens, oddCountPtr, sizeof(int));
			oddCountPtr++;
		}


		origionalElemens++;
	}

}

bool checkEvenOddArrays(arrays evenOddArrays, int* origionaArray, int elementCount){

	int evenCount = 0;
	int oddCount = 0;

	bool elementSplitCorrectly = true;

	for (int i = 0; i < elementCount; i++){

		if (i % 2 == 0){

			if (origionaArray[i] != evenOddArrays.evenArray[evenCount]){

				elementSplitCorrectly = false;
			}
			evenCount++;
		}

		else{

			if (origionaArray[i] != evenOddArrays.oddArray[oddCount]){

				elementSplitCorrectly = false;
			}
			oddCount++;
		}
	}

	return elementSplitCorrectly;
}

__global__ void bitonicSortAllOtherStepsSingleThreaded(int *deviceElements, int step, int phase, int compExchSize, int fullArraySize)
{
	unsigned int  halfstep, halfphase, secondIndex;

	int firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	for (int count = 0; count < fullArraySize / 2; count++){
		halfstep = step / 2;
		halfphase = phase / 2;
		secondIndex = firstIndex^halfstep;

		if ((secondIndex) > firstIndex) {
			if ((firstIndex&halfphase) == 0) {

				if (deviceElements[firstIndex] > deviceElements[secondIndex]) {
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[secondIndex];
					deviceElements[secondIndex] = temp;
				}
			}
			if ((firstIndex&halfphase) != 0) {

				if (deviceElements[firstIndex] < deviceElements[secondIndex]) {
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[secondIndex];
					deviceElements[secondIndex] = temp;
				}
			}
		}
		firstIndex++;
	}
}

/*	As we're complying with the origional model but with arrays reduced in size by two
here we'll just divide firstIndex and secondIndex by two to get our required values.
We're also going to go though all the compaire/exchange operations that would normally be carried out in this step.
*/
__global__ void bitonicSortAllOtherSteps(int *deviceElements, int step, int phase, int compExchSize, int fullArraySize)
{
	unsigned int  halfstep, halfphase, secondIndex;

	int firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	halfstep = step / 2;
	halfphase = phase / 2;
	secondIndex = firstIndex^halfstep;

	if ((secondIndex) > firstIndex) {
		if ((firstIndex&halfphase) == 0) {
			if (deviceElements[firstIndex] > deviceElements[secondIndex]) {
				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}
		if ((firstIndex&halfphase) != 0) {

			if (deviceElements[firstIndex] < deviceElements[secondIndex]) {
				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}

	}
}

__global__ void bitonicSortFirstStep(int *deviceElements, int step, int phase)
{
	unsigned int firstIndex, secondIndex;
	firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	secondIndex = firstIndex^step;

	if ((secondIndex)>firstIndex) {
		if ((firstIndex& phase) == 0) {

			if (deviceElements[firstIndex]>deviceElements[secondIndex]) {
				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}
		if ((firstIndex&phase) != 0) {

			if (deviceElements[firstIndex]<deviceElements[secondIndex]) {

				int temp = deviceElements[firstIndex];
				deviceElements[firstIndex] = deviceElements[secondIndex];
				deviceElements[secondIndex] = temp;
			}
		}
	}
}

void partitioned2dArrayBitonicSort(int *values, int elementCount, int blockCount, int stepOneThreadCount, int allOtherStepsThreadCount)
{
	int *deviceElements;
	int *deviceEvenArray;
	int *deviceOddArray;
	size_t size = elementCount * sizeof(int);

	size_t evenOddSize = elementCount / 2 * sizeof(int);

	arrays evenOddArrays;


	//Allocate half of element size to each of the odd and even arrays
	evenOddArrays.evenArray = (int*)malloc((elementCount)* sizeof(int));
	evenOddArrays.oddArray = (int*)malloc((elementCount)* sizeof(int));

	hipMalloc((void**)&deviceElements, size);
	hipMalloc((void**)&deviceEvenArray, evenOddSize);
	hipMalloc((void**)&deviceOddArray, evenOddSize);

	dim3 blocks(blockCount, 1);
	dim3 stepOneThreads(stepOneThreadCount, 1);

	dim3 allOtherStepThreads(allOtherStepsThreadCount, 1);

	int compExchCount = (elementCount / 4);

	int step, phase;

	for (phase = 2; phase <= elementCount; phase <<= 1) {
		evenOddArrays = splitArray(values, evenOddArrays, elementCount);

		hipMemcpy(deviceEvenArray, evenOddArrays.evenArray, evenOddSize, hipMemcpyHostToDevice);
		hipMemcpy(deviceOddArray, evenOddArrays.oddArray, evenOddSize, hipMemcpyHostToDevice);

		for (step = phase >> 1; step > 0; step = step >> 1) {

			if (step != 1){

				//Even and odd arrays to kernals These two calls to the kernal can run in parallel as there is no cudaSyn device calls between them	
				bitonicSortAllOtherSteps << <blocks, allOtherStepThreads >> >(deviceEvenArray, step, phase, compExchCount, elementCount);
				bitonicSortAllOtherSteps << <blocks, allOtherStepThreads >> >(deviceOddArray, step, phase, compExchCount, elementCount);

			}

			//The last step, so copy back the sorted even odd arrays, merge them into the origional element array copy that to memory then sort it 
			else{

				hipMemcpy(evenOddArrays.evenArray, deviceEvenArray, evenOddSize, hipMemcpyDeviceToHost);
				hipMemcpy(evenOddArrays.oddArray, deviceOddArray, evenOddSize, hipMemcpyDeviceToHost);

				mergeArrays(evenOddArrays, values, elementCount);

				hipMemcpy(deviceElements, values, size, hipMemcpyHostToDevice);
				bitonicSortFirstStep << <blocks, stepOneThreads >> >(deviceElements, step, phase);

				hipMemcpy(values, deviceElements, size, hipMemcpyDeviceToHost);

			}
		}
	}

	hipFree(deviceElements);
	hipFree(deviceEvenArray);
	hipFree(deviceOddArray);

}