#include "hip/hip_runtime.h"

#include "threadPerElementKernel.h"
/*
We get our indexes to swap buy xoring our thread index with the step. This essentially wraps the thread Id round the step value meaning
the only values porduced that are bigger than the Thread Id will be those within the requied step length
*/
__global__ void threadPerElementkernel(int *dev_values, int step, int phaseLength)
{
	unsigned int firstIndex, XoredSecondIndex;
	//Set it to the thread Id
	firstIndex = threadIdx.x + blockDim.x * blockIdx.x;

	XoredSecondIndex = firstIndex ^ step;

	//Threads i corrasponding to the desired bitonic element will be used for the swap
	if ((XoredSecondIndex)>firstIndex) {

		if ((firstIndex&phaseLength) == 0) {
			if (dev_values[firstIndex]>dev_values[XoredSecondIndex]) {

				int temp = dev_values[firstIndex];
				dev_values[firstIndex] = dev_values[XoredSecondIndex];
				dev_values[XoredSecondIndex] = temp;
			}
		}
		if ((firstIndex&phaseLength) != 0) {

			if (dev_values[firstIndex]<dev_values[XoredSecondIndex]) {
				int temp = dev_values[firstIndex];
				dev_values[firstIndex] = dev_values[XoredSecondIndex];
				dev_values[XoredSecondIndex] = temp;
			}
		}
	}

}

/*
Main function call. Created array and calls stepskernel based of the size of the bitonic sequences and step.
*/
void threadPerElementBitonicSortSetup(int *values, int elementsToSort, int deviceBlocks, int threadsPerBlock)
{
	int *dev_values;
	size_t size = elementsToSort* sizeof(int);

	//Allocate memory on the device then copy our host array to device pointer
	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	dim3 blocks(deviceBlocks, 1);
	dim3 threads(threadsPerBlock, 1);

	int step, phaseLength;

	for (phaseLength = 2; phaseLength <= elementsToSort; phaseLength <<= 1) {


		for (step = phaseLength >> 1; step>0; step = step >> 1) {
			threadPerElementkernel << <blocks, threads >> >(dev_values, step, phaseLength);
		}
	}

	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}






